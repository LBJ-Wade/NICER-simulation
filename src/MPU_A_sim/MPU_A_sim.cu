#include "hip/hip_runtime.h"
#include "warpspeed.cuh"
#include <math.h>
#include <stdio.h>

/*
NICER's vital stats.
Note that to use the warpspeed random number generator, SHAPERS_PER_MPU 
must be 32.
*/

// #define NUM_MPU 7
#define NUM_MPU (2*7)
#define DETECTORS_PER_MPU 8
#define SHAPERS_PER_DETECTOR 4
#define SHAPERS_PER_MPU (SHAPERS_PER_DETECTOR * DETECTORS_PER_MPU)
#define SHAPER_ORDER 6

/*
At this level, we have a data vector per thread, so we don't
explicitly identify which detector or shaper the data belongs to.
That's implicit in which vector we're looking at.
*/

struct detector_output {
	int step;
	float charge;
};


struct event_output {
	int rise, fall, zero;
	float sample;
};

/* For the rare case of a fall without a zero crossing. */

#define NO_ZERO 0xffffffff
/* 
Data structures in managed memory. 
*/

enum trigger_state { idle, above, below };

__managed__ struct configuration { 

	float cout[SHAPER_ORDER+1];	/* output weights: include gain here */
	float cback[SHAPER_ORDER];
	float noise;
	float lld;		/* set this to infinity for the unipolar shapers */
	float x[SHAPER_ORDER];
	enum trigger_state state;
	struct detector_output *ip;
	struct event_output *op;
	
} config[NUM_MPU][DETECTORS_PER_MPU][SHAPERS_PER_DETECTOR];

__shared__ float y[SHAPERS_PER_MPU];	/* Shaper outputs are shared within a warp */

__global__ void run_shapers( int steps_to_do, unsigned int *warpspeed_state )
{
	int detector = threadIdx.x/4;
	int shaper = threadIdx.x%4;
	int mpu = blockIdx.x;
	struct configuration *c = &config[mpu][detector][shaper];
	float cout0 = c->cout[0];
	float cout1 = c->cout[1];
	float cout2 = c->cout[2];
	float cout3 = c->cout[3];
	float cout4 = c->cout[4];
	float cout5 = c->cout[5];
	float cout6 = c->cout[6];
	float cback1 = c->cback[0];
	float cback2 = c->cback[1];
	float cback3 = c->cback[2];
	float cback4 = c->cback[3];
	float cback5 = c->cback[4];
	float cback6 = c->cback[5];
	float x1 = c->x[0];
	float x2 = c->x[1];
	float x3 = c->x[2];
	float x4 = c->x[3];
	float x5 = c->x[4];
	float x6 = c->x[5];
	float noise = c->noise;
	float lld = c->lld;
	enum trigger_state state = c->state;
	struct detector_output *ip = c->ip;
	struct event_output *op = c->op;
	
	int step;
	float u, yt, fb;
	float charge = 0.0;
	int next_input = ip->step;
	
	warpspeed_initialize( warpspeed_state );
	
//	mpu == 3 && threadIdx.x == 5 && printf( "Init\n" );
//	mpu == 3 && threadIdx.x == 5 && printf( "steps to do %d\n", steps_to_do );
		
	for( step = 0; step < steps_to_do; step += 1 ) {
//		mpu == 3 && threadIdx.x == 5 && printf( "step %d\n", step );
//		mpu == 3 && threadIdx.x == 5 && printf( "next %d\n", next_input );		
		if( step == next_input ) {
			charge = ip++->charge;
			next_input = ip->step;
		}
		
		u = charge + noise * ( (float) warpspeed_urand() - 2147483648.0 );
		
		y[threadIdx.x] = yt = 
			cout0*u + cout1*x1 + cout2*x2 +cout3*x3 + cout4*x4 + 
			cout5*x5 + cout6*x6;
		fb = u + cback1*x1 + cback2*x2 +cback3*x3 + cback4*x4 + 
			cback5*x5 + cback6*x6;

		x1 = x2;
		x2 = x3;
		x3 = x4;
		x4 = x5;
		x5 = x6;
		x6 = fb;
		
//		mpu == 3 && threadIdx.x == 5 && printf( "sync\n" );
		__syncthreads();
		
		switch( state ) {
			
			case idle:
			if( yt > lld ) {
				state = above;
				op->rise = step;
			}
			break;
			
			case above:
			if( yt < lld ) {
				state = below;
				op->fall = step;
			}
			break;
			
			case below:
			if( yt < 0 ) {
				op->zero = step;
				op++->sample = y[threadIdx.x+1];	/* Unipolar output */
				state = idle;
			}
			else if ( yt > lld ) { 	/* rare double event */
				op++->zero = NO_ZERO;
				op->rise = step;
				state = above;
			}
			break;
			
		}
//		mpu == 3 && threadIdx.x == 5 && printf( "end switch\n" );		
	}
	
	mpu == 0 && threadIdx.x == 0 && printf( "steps done %d\n", step );
		
	warpspeed_save( warpspeed_state );
	
	c->x[0] = x1;
	c->x[1] = x2;
	c->x[2] = x3;
	c->x[3] = x4;
	c->x[4] = x5;
	c->x[5] = x6;
	c->state = state;
	c->op = op;
}

__managed__ struct detector_output null_in[1];
__managed__ struct event_output dummy_out[1];

int main()
{
	struct configuration init;
	unsigned int *random_state = warpspeed_seed( NUM_MPU, 100951 );
	int i, j, k;
	
	for( i = 0; i < SHAPER_ORDER+1; i+=1 ){
		init.cout[i] = 0.1;
	}
	
	for( i = 0; i < SHAPER_ORDER; i+=1 ){
		init.cback[i] = 0.1;
	}
	
	init.noise = 0.001;
	init.lld = INFINITY;
	
	for( i = 0; i < SHAPER_ORDER; i+=1 ){
		init.x[i] = 0.0;
	}
	
	init.state = idle;
	init.ip = null_in;
	init.op = dummy_out;
	
	null_in->step = 2147483647;	/* As big as we can go, better not ask for more steps */
	
	for( i = 0; i < NUM_MPU; i+=1 )
		for( j = 0; j < DETECTORS_PER_MPU; j+=1 )
			for( k = 0; k < SHAPERS_PER_DETECTOR; k+=1 )
				config[i][j][k] = init;
	
	run_shapers<<<NUM_MPU,SHAPERS_PER_MPU>>>( 20000000, random_state);
	hipDeviceSynchronize();
	
}
